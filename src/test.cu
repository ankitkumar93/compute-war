#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void Add(int *a, int *b, int* c)
{
    *c = *a + *b; 
}

int main()
{
    // Host numbers
    int hostA;
    int hostB;
    int hostC;
    
    // Device numbers
    int* devA;
    int* devB;
    int* devC;

    // Allocate memory for device numbers
    hipError_t err = hipMalloc((void**)&devA, sizeof(int));
    if (err != hipSuccess)
    {   
        printf("Failed to alloc memory for A, err: %s\n", hipGetErrorString(err));
    }   

    err = hipMalloc((void**)&devB, sizeof(int));
    if (err != hipSuccess)
    {   
        printf("Failed to alloc memory for B, err: %s\n", hipGetErrorString(err));
    }   
    
    err = hipMalloc((void**)&devC, sizeof(int));
    if (err != hipSuccess)
    {   
        printf("Failed to alloc memory for C, err: %s\n", hipGetErrorString(err));
    }   

    hostA = 10; 
    hostB = 100;

    // Copy host values to device
    hipMemcpy(devA, &hostA, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devB, &hostB, sizeof(int), hipMemcpyHostToDevice);


    // Add on device
    Add<<<1,1>>>(devA, devB, devC);

    // Copy the result back to host
    hipMemcpy(&hostC, devC, sizeof(int), hipMemcpyDeviceToHost);

    // Deallocate memory
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    printf("A: %d, B: %d, C = A + B: %d\n", hostA, hostB, hostC);

    return 0;
}
